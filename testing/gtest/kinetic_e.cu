#include <thrust/iterator/counting_iterator.h>
#include <thrusting/vector.h>

#include <bphcuda/kinetic_e.h>

#include <gtest/gtest.h>

namespace {
  using thrusting::real;
}

TEST(kinetic_e, test1){
  real e = bphcuda::calc_kinetic_e(
    1,
    thrust::make_counting_iterator<real>(-1),    
    thrust::make_counting_iterator<real>(2),    
    thrust::make_counting_iterator<real>(3),    
    thrust::make_counting_iterator<real>(2)); // m

  EXPECT_EQ(14.0, e);
}
