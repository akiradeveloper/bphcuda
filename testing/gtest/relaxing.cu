#include <iostream>

#include <thrusting/dtype/real.h>
#include <thrusting/vector.h>
#include <thrusting/iterator.h>

#include <bphcuda/relaxing.h>

#include <gtest/gtest.h>

namespace {
  using thrusting::real;
  using thrusting::real3;
}

// case 3 particles 
TEST(relaxing, n_particle_even){
  size_t n_particle = 3;
  real _us[] = {1.0, 4.0, 7.0}; THRUSTING_VECTOR<real> us(_us, _us+n_particle);
  real _vs[] = {2.0, 5.0, 8.0}; THRUSTING_VECTOR<real> vs(_vs, _vs+n_particle);
  real _ws[] = {3.0, 6.0, 9.0}; THRUSTING_VECTOR<real> ws(_ws, _ws+n_particle);

  real mass = 1.0;

  real3 new_momentum =
    bphcuda::calc_momentum(
      n_particle,
      us.begin(),
      vs.begin(),
      ws.begin(),
      thrust:constant_iterator<real>(mass));

  real old_kinetic_e =
    bphcuda::calc_kinetic_e(
      n_particle,
      us.begin(),
      vs.begin(),
      ws.begin(),
      thrust::constant_iterator<real>(mass));
  
  size_t seed = 0;
  bphcuda::relax(
    n_particle,
    us.begin(),
    vs.begin(),
    ws.begin(),
    seed);

  real3 new_momentum =
    bphcuda::calc_momentum(
      n_particle,
      us.begin(),
      vs.begin(),
      ws.begin(),
      thrust:constant_iterator<real>(mass));
    
  real new_kinetic_e =
    bphcuda::calc_kinetic_e(
      n_particle,
      us.begin(),
      vs.begin(),
      ws.begin(),
      thrust::constant_iterator<real>(mass));
  
  // the last element is zero speed
  EXPECT_EQ(
    real3(0.0, 0.0, 0.0),
    thrusting::iterator_value_at(2, thrusting::make_zip_iterator(us.begin(), vs.begin(), ws.begin())));
  // preserving the momentum
  EXPECT_EQ(old_momentum, new_momentum);
  // preserving the energy
  EXPECT_EQ(old_kinetic_e, new_kinetic_e);
}

// case 2 particles
TEST(relaxing, n_particle_odd){
}
